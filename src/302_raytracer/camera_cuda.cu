/**
 * @file camera_cuda.cu
 * @brief CUDA-accelerated ray tracing template
 */

#include <cfloat>
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

//==============================================================================
// VECTOR MATH AND UTILITY STRUCTURES
//==============================================================================

/**
 * @brief Simple 3D vector structure optimized for CUDA
 * Provides basic vector operations for ray tracing computations
 */
struct float3_simple
{
   float x, y, z;
   __device__ __host__ float3_simple() : x(0), y(0), z(0) {}
   __device__ __host__ float3_simple(float x_, float y_, float z_) : x(x_), y(y_), z(z_) {}

   __device__ __host__ float3_simple operator+(const float3_simple &other) const
   {
      return float3_simple(x + other.x, y + other.y, z + other.z);
   }

   __device__ __host__ float3_simple operator-(const float3_simple &other) const
   {
      return float3_simple(x - other.x, y - other.y, z - other.z);
   }

   __device__ __host__ float3_simple operator*(float t) const { return float3_simple(x * t, y * t, z * t); }

   __device__ __host__ float3_simple operator/(float t) const { return float3_simple(x / t, y / t, z / t); }

   __device__ __host__ float3_simple operator-() const { return float3_simple(-x, -y, -z); }

   __device__ __host__ float length() const { return sqrtf(x * x + y * y + z * z); }

   __device__ __host__ float length_squared() const { return x * x + y * y + z * z; }
};

__device__ __host__ float3_simple operator*(float t, const float3_simple &v) { return v * t; }

/** @brief Compute dot product of two vectors */
__device__ __host__ float dot(const float3_simple &a, const float3_simple &b)
{
   return a.x * b.x + a.y * b.y + a.z * b.z;
}

/** @brief Compute cross product of two vectors */
__device__ __host__ float3_simple cross(const float3_simple &a, const float3_simple &b)
{
   return float3_simple(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

/** @brief Normalize a vector to unit length */
__device__ __host__ float3_simple unit_vector(const float3_simple &v) { return v / v.length(); }

//==============================================================================
// RAY TRACING DATA STRUCTURES AND FUNCTIONS
//==============================================================================

/**
 * @brief Simple ray structure for ray tracing calculations
 */
struct ray_simple
{
   float3_simple orig, dir; ///< Ray origin and direction

   __device__ __host__ ray_simple() {}
   __device__ __host__ ray_simple(const float3_simple &origin, const float3_simple &direction)
       : orig(origin), dir(direction)
   {
   }

   /** @brief Get point along ray at parameter t */
   __device__ __host__ float3_simple at(float t) const { return orig + t * dir; }
};

/** @brief Convert a normal to a debug RGB color */
__device__ __host__ inline float3_simple normal_to_color(const float3_simple &n)
{
   return float3_simple(0.5f * (n.x + 1.0f), 0.5f * (n.y + 1.0f), 0.5f * (n.z + 1.0f));
}

//==============================================================================
// RANDOM NUMBER GENERATION AND SAMPLING
//==============================================================================

/** @brief Generate random float in range [0,1) using CUDA's curand */
__device__ float random_float(hiprandState *state) { return hiprand_uniform(state); }

/** @brief Smooth interpolation function for gradual transitions */
__device__ float smoothstep(float edge0, float edge1, float x)
{
   float t = fmaxf(0.0f, fminf(1.0f, (x - edge0) / (edge1 - edge0)));
   return t * t * (3.0f - 2.0f * t);
}

//==============================================================================
// CUDA KERNELS
//==============================================================================

/**
 * @brief Initialize random states for all threads
 * This kernel should be called once at startup to initialize the shared random state array
 * @param rand_states Array of random states (one per thread/pixel)
 * @param num_states Total number of states to initialize
 * @param seed Base seed for random number generation
 */
__global__ void init_random_states(hiprandState *rand_states, int num_states, unsigned long long seed)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx < num_states)
   {
      // Initialize each state with a unique seed based on index
      hiprand_init(seed + idx, 0, 0, &rand_states[idx]);
   }
}

/**
 * @brief Main CUDA kernel for ray tracing entire image
 * Each thread processes one pixel with multiple samples for anti-aliasing
 * @param image Output image buffer (RGB, 8-bit per channel)
 * @param width Image width in pixels
 * @param height Image height in pixels
 * @param samples_per_pixel Number of rays per pixel for anti-aliasing
 * @param max_depth Maximum ray recursion depth
 * @param cam_center_* Camera center position components
 * @param pixel00_* Top-left pixel center position components
 * @param delta_u_* Pixel step in U direction components
 * @param delta_v_* Pixel step in V direction components
 * @param ray_count Global counter for rays traced
 * @param rand_states Shared array of random states (one per thread/pixel)
 */
__global__ void renderKernel(unsigned char *image, int width, int height, int samples_per_pixel, int max_depth,
                             float cam_center_x, float cam_center_y, float cam_center_z, float pixel00_x,
                             float pixel00_y, float pixel00_z, float delta_u_x, float delta_u_y, float delta_u_z,
                             float delta_v_x, float delta_v_y, float delta_v_z, hiprandState *rand_states)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;

   // Strict bounds checking
   if (x >= width || y >= height)
      return;

   int pixel_idx = y * width + x;
   int base_idx = pixel_idx * 3;

   // Double check bounds for memory access
   if (pixel_idx >= width * height || base_idx + 2 >= width * height * 3)
   {
      return;
   }

   // Use the pre-initialized random state for this pixel
   hiprandState *local_rand_state = &rand_states[pixel_idx];

   // Convert parameters to float3_simple
   float3_simple camera_center(cam_center_x, cam_center_y, cam_center_z);
   float3_simple pixel_color(0, 0, 0);

   pixel_color = pixel_color + float3_simple(random_float(local_rand_state), random_float(local_rand_state),
                                             random_float(local_rand_state));

   // Gamma correction (gamma=2)
   pixel_color.x = sqrtf(fmaxf(pixel_color.x, 0.0f));
   pixel_color.y = sqrtf(fmaxf(pixel_color.y, 0.0f));
   pixel_color.z = sqrtf(fmaxf(pixel_color.z, 0.0f));

   // Convert to bytes with clamping
   unsigned char r = (unsigned char)(255.0f * fminf(fmaxf(pixel_color.x, 0.0f), 1.0f));
   unsigned char g = (unsigned char)(255.0f * fminf(fmaxf(pixel_color.y, 0.0f), 1.0f));
   unsigned char b = (unsigned char)(255.0f * fminf(fmaxf(pixel_color.z, 0.0f), 1.0f));

   // Store in image buffer - each kernel writes to its own unique location
   image[base_idx] = r;
   image[base_idx + 1] = g;
   image[base_idx + 2] = b;
}

//==============================================================================
// HOST INTERFACE FUNCTIONS
//==============================================================================

/**
 * @brief Host function for tile-based rendering (useful for real-time display)
 * Renders only a rectangular portion of the image for progressive rendering
 * @param image Full image buffer (input/output)
 * @param width Full image width in pixels
 * @param height Full image height in pixels
 * @param cam_center_* Camera position components
 * @param pixel00_* Top-left pixel center position components
 * @param delta_u_* Pixel step in U direction components
 * @param delta_v_* Pixel step in V direction components
 * @param samples_per_pixel Number of rays per pixel for anti-aliasing
 * @param max_depth Maximum ray recursion depth
 */
extern "C" unsigned long long renderPixelsCUDA(unsigned char *image, int width, int height, double cam_center_x,
                                               double cam_center_y, double cam_center_z, double pixel00_x,
                                               double pixel00_y, double pixel00_z, double delta_u_x, double delta_u_y,
                                               double delta_u_z, double delta_v_x, double delta_v_y, double delta_v_z,
                                               int samples_per_pixel, int max_depth)
{

   // Allocate device memory for the full image (we need to maintain the full buffer)
   unsigned char *d_image;
   size_t image_size = width * height * 3 * sizeof(unsigned char);
   int num_pixels = width * height;

   // Random generato states
   hiprandState *d_rand_states;

   hipError_t malloc_err1 = hipMalloc(&d_image, image_size);
   hipError_t malloc_err2 = hipMalloc(&d_rand_states, num_pixels * sizeof(hiprandState));

   if (malloc_err1 != hipSuccess || malloc_err2 != hipSuccess)
   {
      printf("CUDA malloc error: %s, %s\n", hipGetErrorString(malloc_err1), hipGetErrorString(malloc_err2));
      return 0;
   }

   // Initialize random states for all pixels
   int threads_per_block = 256;
   int num_blocks = (num_pixels + threads_per_block - 1) / threads_per_block;
   init_random_states<<<num_blocks, threads_per_block>>>(d_rand_states, num_pixels, 1984);

   hipError_t init_err = hipGetLastError();
   if (init_err != hipSuccess)
   {
      printf("CUDA random state init error: %s\n", hipGetErrorString(init_err));
      hipFree(d_image);
      hipFree(d_rand_states);
      return 0;
   }

   hipDeviceSynchronize();

   // Set up grid and block dimensions for the tile
   dim3 block_size(32, 4);
   dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);

   printf("Tile grid size: (%d, %d), Block size: (%d, %d)\n", grid_size.x, grid_size.y, block_size.x, block_size.y);

   // Launch tile rendering kernel
   renderKernel<<<grid_size, block_size>>>(d_image, width, height, samples_per_pixel, max_depth, (float)cam_center_x,
                                           (float)cam_center_y, (float)cam_center_z, (float)pixel00_x, (float)pixel00_y,
                                           (float)pixel00_z, (float)delta_u_x, (float)delta_u_y, (float)delta_u_z,
                                           (float)delta_v_x, (float)delta_v_y, (float)delta_v_z, d_rand_states);

   // Check for kernel errors
   hipError_t kernel_err = hipGetLastError();
   if (kernel_err != hipSuccess)
   {
      printf("CUDA kernel error: %s\n", hipGetErrorString(kernel_err));
      hipFree(d_image);
      return 0;
   }

   hipDeviceSynchronize();

   // Copy result back to host
   hipError_t copy_err = hipMemcpy(image, d_image, image_size, hipMemcpyDeviceToHost);
   if (copy_err != hipSuccess)
   {
      printf("Memory copy error: %s\n", hipGetErrorString(copy_err));
      hipFree(d_image);
      return 0;
   }

   // Clean up
   hipFree(d_image);

   return 1;
}